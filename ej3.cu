#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hiprand.h"

// uint_to_int_conversion
// transforms unsigned int to int between 0 and n - 1
__global__ void uint_to_int_conversion(unsigned int *input, int *output, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	output[i] = input[i] % n;
}

// se asume que el tamaño de perm es igual al del bloque
// y que las premutaciones son válidas
__global__ void block_perm(int *data, int *perm, int length) {
	int off = blockIdx.x * blockDim.x;
	__shared__ int shared_data[1024];

	if (length < off + threadIdx.x) return;

	shared_data[threadIdx.x] = data[off];
	__syncthreads();

	data[off+threadIdx.x] = shared_data[perm[threadIdx.x]];
}

__global__ void block_perm_org(int * data, int *perm, int length) {
	int off = blockIdx.x * blockDim.x;

	if (length < off+threadIdx.x) return;

	int perm_data = data[off + perm[threadIdx.x]];

	__syncthreads();

	data[off + threadIdx.x] = perm_data;
}

int main(int argc, char *argv[]) {
	int *data, *perm;
	unsigned int *uperm;

	if (argc < 3) {
		printf("Usage: %s <data_length> <variant>\n", argv[0]);
		return 1;
	}

	int length = atoi(argv[1]);
	length = length * length;
	int variant = atoi(argv[2]);

	hipMalloc(&data, sizeof(int) * length);
	hipMalloc(&perm, sizeof(int) * 1024);

	hipMemset(data, 0, sizeof(int) * length);
	// hipMemset(perm, 0, sizeof(int) * 1024);

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
	hiprandGeneratePoisson(gen, uperm, 1024, 4.0);
	hiprandDestroyGenerator(gen);

	uint_to_int_conversion<<<1, 1024>>>(uperm, perm, 1024);

	// Fill perm with random int
	// generator<<<1, 1024>>>(1024, perm, length);

	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(length / 1024, 1, 1);

	if (variant == 0) {
		block_perm_org<<<dimGrid, dimBlock>>>(data, perm, length);
	} else {
		block_perm<<<dimGrid, dimBlock>>>(data, perm, length);
	}

	hipFree(data);
	hipFree(perm);
}
