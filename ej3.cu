
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

// se asume que el tamaño de perm es igual al del bloque
// y que las premutaciones son válidas
__global__ void block_perm(int *data, int *perm, int length) {
	int off = blockIdx.x * blockDim.x;
	__shared__ int *shared_pem;
	int perm_data;

	if (length < off + threadIdx.x) return;

	shared_pem = &perm[threadIdx.x];
	__syncwarp();
	perm_data = data[off + *shared_pem];
	__syncthreads();

	data[off+threadIdx.x] = perm_data;
}

__global__ void block_perm_org(int * data, int *perm, int length) {
	int off = blockIdx.x * blockDim.x;

	if (length < off+threadIdx.x) return;

	int perm_data = data[off + perm[threadIdx.x]];

	__syncthreads();

	data[off + threadIdx.x] = perm_data;
}

int main(int argc, char *argv[]) {
	int *data, *perm;

	if (argc < 3) {
		printf("Usage: %s <data_length> <variant>\n", argv[0]);
		return 1;
	}

	int length = atoi(argv[1]);
	length = length * length;
	int variant = atoi(argv[2]);

	hipMalloc(&data, sizeof(int) * length);
	hipMalloc(&perm, sizeof(int) * length);

	hipMemset(data, 0, sizeof(int) * length);
	hipMemset(perm, 0, sizeof(int) * length);

	dim3 dimBlock(256, 1, 1);
	dim3 dimGrid(length / 256, 1, 1);

	if (variant == 0) {
		block_perm_org<<<dimGrid, dimBlock>>>(data, perm, length);
	} else {
		block_perm<<<dimGrid, dimBlock>>>(data, perm, length);
	}

	hipFree(data);
	hipFree(perm);
}
