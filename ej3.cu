#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void generator(int num_points, int *points, int max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < num_points) {
		// sudo random generator on device
    points[i] = max(i, max) >> 2 + i;
  }
}


// se asume que el tamaño de perm es igual al del bloque
// y que las premutaciones son válidas
__global__ void block_perm(int *data, int *perm, int length) {
	int off = blockIdx.x * blockDim.x;
	// int shared_pem;
	__shared__ int shared_data[1024];

	if (length < off + threadIdx.x) return;

	// shared_pem[threadIdx.x] = perm[threadIdx.x];
	shared_data[threadIdx.x] = data[off + threadIdx.x];
	__syncthreads();

	data[off+threadIdx.x] = shared_data[perm[threadIdx.x]];
}

__global__ void block_perm_org(int * data, int *perm, int length) {
	int off = blockIdx.x * blockDim.x;

	if (length < off+threadIdx.x) return;

	int perm_data = data[off + perm[threadIdx.x]];

	__syncthreads();

	data[off + threadIdx.x] = perm_data;
}

int main(int argc, char *argv[]) {
	int *data, *perm;

	if (argc < 3) {
		printf("Usage: %s <data_length> <variant>\n", argv[0]);
		return 1;
	}

	int length = atoi(argv[1]);
	length = length * length;
	int variant = atoi(argv[2]);

	hipMalloc(&data, sizeof(int) * length);
	hipMalloc(&perm, sizeof(int) * 1024);

	hipMemset(data, 0, sizeof(int) * length);

	// Fill perm with random int
	generator<<<1, 1024>>>(1024, perm, length);

	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(length / 1024, 1, 1);

	if (variant == 0) {
		block_perm_org<<<dimGrid, dimBlock>>>(data, perm, length);
	} else {
		block_perm<<<dimGrid, dimBlock>>>(data, perm, length);
	}

	hipFree(data);
	hipFree(perm);
}
