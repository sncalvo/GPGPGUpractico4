#include "hip/hip_runtime.h"
#include "./common.h"
#include "./generator.cuh"

__global__ void generator(unsigned int num_points, double *points) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < num_points && j < num_points) {
    points[i * num_points + j] = i + j;
  }
}

/*
  special_sum calculates the sum of all values inside a matrix in a radius.

  num_points: number of points in a matrix
  sum_result: pointer to the result array
  radius: radius of elements to sum
  matrix: pointer to the matrix with values
*/
__global__ void special_sum(unsigned int num_points, double *sum_result, int radius, double *matrix) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ double matrix_point[32][32];

  if (i >= num_points || j >= num_points) {
    return;
  }

  unsigned int threadIdx_x = threadIdx.x + 2;
  unsigned int threadIdx_y = threadIdx.y + 2;

  matrix_point[threadIdx_y -1][threadIdx_x -1] = matrix[i * num_points + j];
  __syncwarp();

  double result = -2 * radius * matrix_point[threadIdx_y][threadIdx_x];

  for (int offset = -radius; offset <= radius; offset++) {
    result += matrix_point[threadIdx_y + offset][threadIdx_x];
    result += matrix_point[threadIdx_y][threadIdx_x + offset];
  }

  sum_result[j + i * num_points] = result / SMALL_POINT_SIZE;
}


// calculate_sin: Calculates sin of x and y coordinates for points inside a square
__global__ void calculate_sin(unsigned int num_points, Point2D *points, double *sin_result) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ Point2D *point;
  point = &points[j + i * num_points];

  double x = point->x;
  double y = point->y;

  sin_result[j + i * num_points] = sin(x + y);
}

int main(int argc, char *argv[]) {
	int num_points_2d = 0;

	if (argc < 2) {
		printf("Debe ingresar la cantidad de puntos\n");
		return 0;
	} else {
		num_points_2d = atoi(argv[1]);
	}

  // Builds square of points with space of SMALL_POINT_SIZE
  // unsigned int num_points_2d = trunc(SQUARE_LENGTH / SMALL_POINT_SIZE); // 6_280

  size_t size_2d = num_points_2d * num_points_2d * sizeof(double); // 39_438_400 x double_size

  double *d_points_2d;
  CUDA_CHK(hipMalloc((void**)&d_points_2d, size_2d));

  // dim3 block_dim(32, 32, 1);
  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid_dim(num_points_2d / BLOCK_SIZE, num_points_2d / BLOCK_SIZE);

  // Generates points
  generator<<<grid_dim, block_dim>>>(num_points_2d * num_points_2d, d_points_2d);
  CUDA_CHK(hipGetLastError());
  CUDA_CHK(hipDeviceSynchronize());

  // // Generates points inside a square
  // generate_square<<<grid_dim, block_dim>>>(d_points_2d, num_points_2d);
  // CUDA_CHK(hipGetLastError());

  // CUDA_CHK(hipDeviceSynchronize());

  // // Point3D *points_3d = (Point3D *)malloc(num_points_3d * num_points_3d * num_points_3d * sizeof(Point3D));
  // // CUDA_CHK(hipMemcpy(points_3d, d_points_3d, num_points_3d * num_points_3d * num_points_3d * sizeof(Point3D), hipMemcpyDeviceToHost));

  // // Calculates sin of points
  // double *d_sin_result;
  // CUDA_CHK(hipMalloc((void **)&d_sin_result, num_points_2d * num_points_2d * sizeof(double)));

  // calculate_sin<<<grid_dim, block_dim>>>(num_points_2d, d_points_2d, d_sin_result);
  // CUDA_CHK(hipGetLastError());
  // CUDA_CHK(hipDeviceSynchronize());

  double *gpu_special_sum_result;
  CUDA_CHK(hipMalloc((void **)&gpu_special_sum_result, num_points_2d * num_points_2d * sizeof(double)));

  special_sum<<<grid_dim, block_dim>>>(num_points_2d, gpu_special_sum_result, 1, d_points_2d);
  CUDA_CHK(hipGetLastError());
  CUDA_CHK(hipDeviceSynchronize());

  double *special_sum_result = (double *)malloc(num_points_2d * num_points_2d * sizeof(double));
  CUDA_CHK(hipMemcpy(special_sum_result, gpu_special_sum_result, num_points_2d * num_points_2d * sizeof(double), hipMemcpyDeviceToHost));

  // print_matrix_of_points(special_sum_result, 64);

  free(special_sum_result);
  // CUDA_CHK(hipFree(d_sin_result));
  CUDA_CHK(hipFree(d_points_2d));
  CUDA_CHK(hipFree(gpu_special_sum_result));

  return 0;
}
