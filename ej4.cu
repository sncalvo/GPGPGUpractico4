#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

#define TSZ 32
#define DATA_SIZE 128

__global__ void sum_col_block(int *data, int length) {
	__shared__ int sh_tile[TSZ][TSZ];

	int n = gridDim.x * blockDim.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	sh_tile[threadIdx.y][threadIdx.x] = data[idy*n+idx];

	__syncthreads();

	int col_sum = sh_tile[threadIdx.x][threadIdx.y];

	for (int i=16; i>0; i/=2)
		col_sum += __shfl_down_sync(0xFFFFFFFF, col_sum, i);

	data[idy*n+idx] = col_sum;
}

int main() {
	int *data_host = (int*)malloc(sizeof(int)*DATA_SIZE*DATA_SIZE);
	int *data;
	hipMalloc((void **)&data, sizeof(int)*DATA_SIZE*DATA_SIZE);

	for (int i=0; i<DATA_SIZE*DATA_SIZE; i++)
		data_host[i]=i;

	hipMemcpy(data, data_host, sizeof(int)*DATA_SIZE*DATA_SIZE, hipMemcpyHostToDevice);

	printf("Has not failed 0 \n");
	fflush(stdout);

	dim3 dimBlock(TSZ, TSZ);
	dim3 dimGrid(DATA_SIZE/TSZ, DATA_SIZE/TSZ);

	printf("Has not failed 1 \n");
	fflush(stdout);

	sum_col_block<<<dimGrid, dimBlock>>>(data, DATA_SIZE*DATA_SIZE);
	hipDeviceSynchronize();
	hipMemcpy(data_host, data, sizeof(int)*DATA_SIZE*DATA_SIZE, hipMemcpyDeviceToHost);

	printf("Has not failed 2 \n");
	fflush(stdout);

	printf("Has not failed 3 \n");
	fflush(stdout);

	for (int i=0; i<DATA_SIZE*DATA_SIZE; i++)
		printf("%d ", data_host[i]);

	hipFree(data);

	free(data_host);

	return 0;
}
