#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <locale.h>

#include "hip/hip_runtime.h"

#define M 256
#define BLOCK_SIZE 1024
#define BLOCK_PROCESS_SIZE 256

#define A 15
#define B 27
#define M 256
#define A_MMI_M -17

#define N 512

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void read_file(const char*, int*);
int get_text_length(const char * fname);

__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void decrypt_kernel(int *d_message, int length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		d_message[i] = modulo(A_MMI_M * (d_message[i] - B), M);
	}
}

__global__ void shared_count_occurences(int *d_message, int occurenses[M], int length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int block = i * BLOCK_PROCESS_SIZE;

	extern __shared__ int shared_message[]; // blockDim * sizeof(int) bytes
	int local_occurenses[256]; // blockDim * sizeof(int) bytes

	for (int j = 0; j < BLOCK_PROCESS_SIZE; j++)
	{
		if (j >= length) {
			break;
		}

		shared_message[threadIdx.x * BLOCK_PROCESS_SIZE + j] = d_message[block + j];
		__syncwarp();
		int occurense_index = modulo(shared_message[threadIdx.x * BLOCK_PROCESS_SIZE + j], M);
		local_occurenses[occurense_index]++;
	}

	for (int j = 0; j < 256; j++)
	{
		atomicAdd(&occurenses[j], local_occurenses[j]);
		__syncthread();
	}
}

__global__ void count_occurences(int *d_message, int occurenses[M], int length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int occurense_index = modulo(d_message[i], M);

	if (i < length)
	{
		atomicAdd(&occurenses[occurense_index], 1);
		// occurenses[modulo(d_message[i], M)]++;
		// __syncthreads();
	}
}

int parte_2(int length, unsigned int size, int *message, int *occurenses)
{
	int *d_message;
	hipMalloc((void**)&d_message, length * sizeof(int));
	hipMemcpy(d_message, message, length * sizeof(int), hipMemcpyHostToDevice);

	int *d_occurenses;
	hipMalloc((void**)&d_occurenses, M * sizeof(int));
	hipMemset(d_occurenses, 0, M * sizeof(int));

	dim3 block_dim(BLOCK_SIZE);
 	dim3 grid_dim(size / block_dim.x);

	decrypt_kernel<<<grid_dim, block_dim>>>(d_message, length);
	// count_occurences<<<grid_dim, block_dim, BLOCK_SIZE * sizeof(int)>>>(d_message, d_occurenses, length);
	grid_dim = dim3(size / (block_dim.x * BLOCK_PROCESS_SIZE));
	shared_count_occurences<<<grid_dim, block_dim, BLOCK_SIZE * BLOCK_PROCESS_SIZE * sizeof(int)>>>(d_message, d_occurenses, length);

	hipMemcpy(message, d_message, size, hipMemcpyDeviceToHost);
	hipMemcpy(occurenses, d_occurenses, M * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_message);
	hipFree(d_occurenses);

	return 0;
}

void print_occurences(int *occurenses)
{
	for (int i = 0; i < 256; i++)
	{
		printf("%d: %d\n", i, occurenses[i]);
	}
}

void print_message(int *message, int length)
{
	for (int i = 0; i < 2048; i++)
	{
		printf("%c", (char)message[i]);
	}
	printf("\n");
}

int main(int argc, char *argv[])
{
	int *h_message;
	// int *d_message;
	unsigned int size;

	const char *fname;

	if (argc < 2) {
		printf("Debe ingresar el nombre del archivo\n");
	} else {
		fname = argv[1];
	}

	int length = get_text_length(fname);

	size = length * sizeof(int);

	// reservar memoria para el mensaje
	h_message = (int *)malloc(size);

	// leo el archivo de la entrada
	read_file(fname, h_message);

	int *h_occurenses = (int *)malloc(M * sizeof(int));

	parte_2(length, size, h_message, h_occurenses);

	print_occurences(h_occurenses);
	free(h_occurenses);

	return 0;
}

int get_text_length(const char * fname)
{
	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags

	size_t pos = ftell(f);
	fseek(f, 0, SEEK_END);
	size_t length = ftell(f);
	fseek(f, pos, SEEK_SET);

	fclose(f);

	return length;
}

void read_file(const char * fname, int* input)
{
	// printf("leyendo archivo %s\n", fname );

	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags
	if (f == NULL) {
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	//fread(input, 1, N, f);
	int c;
	while ((c = getc(f)) != EOF) {
		*(input++) = c;
	}

	fclose(f);
}
