#include <stdio.h>
#include <stdlib.h>

#include <locale.h>

#include "hip/hip_runtime.h"

#define M 256
#define BLOCK_SIZE 1024

#define A 15
#define B 27
#define M 256
#define A_MMI_M -17

#define N 512

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void read_file(const char*, int*);
int get_text_length(const char * fname);

__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void decrypt_kernel(int *d_message, int length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		d_message[i] = modulo(A_MMI_M * (d_message[i] - B), M);
	}
}

__global__ void shared_count_occurences(int *d_message, int occurenses[M], int length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int shared_occurenses[]; // blockDim * sizeof(int) bytes

	if (i > length)
	{
		return;
	}

	int occurense_index = modulo(d_message[i], M);

	atomicAdd(&shared_occurenses[occurense_index], 1);

	__syncthreads();

	atomicAdd(&occurenses[i], shared_occurenses[occurense_index]);
}

__global__ void count_occurences(int *d_message, int occurenses[M], int length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		occurenses[modulo(d_message[i], M)]++;
		__syncthreads();
	}
}

int parte_2(int length, unsigned int size, int *message, int *occurenses)
{
	int *d_message;
	hipMalloc((void**)&d_message, length * sizeof(int));
	hipMemcpy(d_message, message, length * sizeof(int), hipMemcpyHostToDevice);

	int *d_occurenses;
	hipMalloc((void**)&d_occurenses, M * sizeof(int));
	hipMemset(d_occurenses, 0, M * sizeof(int));

	int block_dim = BLOCK_SIZE;
 	int grid_dim = (size + block_dim - 1) / block_dim;

	decrypt_kernel<<<grid_dim, block_dim>>>(d_message, length);
	// count_occurences<<<grid_dim, block_dim, BLOCK_SIZE * sizeof(int)>>>(d_message, d_occurenses, length);
	count_occurences<<<grid_dim, block_dim>>>(d_message, d_occurenses, length);

	hipMemcpy(message, d_message, size, hipMemcpyDeviceToHost);
	hipMemcpy(occurenses, d_occurenses, M * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_message);
	hipFree(d_occurenses);

	return 0;
}

void print_occurences(int *occurenses)
{
	for (int i = 0; i < 256; i++)
	{
		printf("%d: %d\n", i, occurenses[i]);
	}
}

void print_message(int *message, int length)
{
	for (int i = 0; i < 256; i++)
	{
		printf("%c", (char)message[i]);
	}
	printf("\n");
}

int main(int argc, char *argv[])
{
	int *h_message;
	// int *d_message;
	unsigned int size;

	const char *fname;

	if (argc < 2) {
		printf("Debe ingresar el nombre del archivo\n");
	} else {
		fname = argv[1];
	}

	int length = get_text_length(fname);

	size = length * sizeof(int);

	// reservar memoria para el mensaje
	h_message = (int *)malloc(size);

	// leo el archivo de la entrada
	read_file(fname, h_message);

	print_message(h_message, length);

	int *h_occurenses = (int *)malloc(M * sizeof(int));

	parte_2(length, size, h_message, h_occurenses);

	print_message(h_message, length);

	print_occurences(h_occurenses);
	free(h_occurenses);

	return 0;
}

int get_text_length(const char * fname)
{
	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags

	size_t pos = ftell(f);
	fseek(f, 0, SEEK_END);
	size_t length = ftell(f);
	fseek(f, pos, SEEK_SET);

	fclose(f);

	return length;
}

void read_file(const char * fname, int* input)
{
	// printf("leyendo archivo %s\n", fname );

	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags
	if (f == NULL) {
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	//fread(input, 1, N, f);
	int c;
	while ((c = getc(f)) != EOF) {
		*(input++) = c;
	}

	fclose(f);
}
